#include "hip/hip_runtime.h"
// FEITO POR:
// André Miguel Sikora Marques
// GRR20182593

#include <stdlib.h>
#include <stdio.h>
#include <math.h>
#include <float.h>

#include "chrono.c"

#include "hip/hip_runtime_api.h" // mudado aqui por WZ
/**
 * CUDA Kernel Device code
 *
 * Computes the vector addition of A and B into C. The 3 vectors have the same
 * number of elements numElements.
 */

// (c) by W.Zola set/23
//
// EXAMPLE Sequential MAX-Heap,
//				decreaseMax operation
//				and others

// para compilar:
// gcc -O3 max-heap.c -o max-heap -lm

// #define SHOW_DECREASE_MAX_STEPS 1

#define MAX_SIZE ((256 * 1024 * 1024) + 4)
// #define MAX_SIZE 268435460

__global__ void
globalHisto_GM(unsigned int *H, int h, float *Input, int nElements, float nMin, float nMax)
{
	// h <- linha de comando
	// <nBlocks (56), nThreads (1024)>

	int thread_index = threadIdx.x + blockIdx.x * blockDim.x;

	int increment = blockDim.x * gridDim.x;

	float A = nMax - nMin;
	float AF = A / h;
	int faixa;
	for (int i = thread_index; i < nElements; i += increment)
	{
		faixa = (Input[i] - nMin) / AF;
		atomicAdd(&(H[faixa]), 1);
	}
}

__global__ void
blockHisto_SM(unsigned int (*HH)[(48 * 1024) / 4], int h, float *Input, int nElements, float nMin, float nMax)
{
	// h <- linha de comando
	// <nBlocks (56), nThreads (1024)>
	//

	__shared__ unsigned int HS[(48 * 1024) / 4];

	for (int i = threadIdx.x; i < h; i += blockDim.x)
		HS[i] = 0;

	__syncthreads();

	int thread_index = threadIdx.x + blockIdx.x * blockDim.x;

	int increment = blockDim.x * gridDim.x;

	float A = nMax - nMin;
	float AF = A / h;
	int faixa;
	for (int i = thread_index; i < nElements; i += increment)
	{
		faixa = (Input[i] - nMin) / AF;
		atomicAdd(&(HS[faixa]), 1);
	}

	__syncthreads();

	for (int i = threadIdx.x; i < h; i += blockDim.x)
		HH[blockIdx.x][i] += HS[i];
}

__global__ void
globalHisto_SM(unsigned int *H, int h, float *Input, int nElements, float nMin, float nMax)
{
	__shared__ unsigned int HS[(48 * 1024) / 4];

	for (int i = threadIdx.x; i < h; i += blockDim.x)
		HS[i] = 0;

	__syncthreads();

	int thread_index = threadIdx.x + blockIdx.x * blockDim.x;

	int increment = blockDim.x * gridDim.x;

	float A = nMax - nMin;
	float AF = A / h;
	int faixa;
	for (int i = thread_index; i < nElements; i += increment)
	{
		faixa = (Input[i] - nMin) / AF;
		atomicAdd(&(HS[faixa]), 1);
	}

	__syncthreads();

	for (int i = threadIdx.x; i < h; i += blockDim.x)
		atomicAdd(&(H[i]), HS[i]);
}

//////////////
int main(int argc, char *argv[])
{
	// testar com os mesmos valores do trabalho passado

	if (argc < 4)
	{
		fprintf(stderr, "usage:\n\t./trab2 <nTotalElements> <h> <-gh OR -bh OR -ghgm>\n");
		fprintf(stderr, "\t-gh = globalHisto_SM\n");
		fprintf(stderr, "\t-bh = blockHisto_SM\n");
		fprintf(stderr, "\t-ghgm = globalHisto_GM\n");
		exit(-1);
	}

	//printf("%s\n%s\n%s\n%s\n", argv[0], argv[1], argv[2], argv[3]);

	int nTotalElements = atoi(argv[1]);
	int h = atoi(argv[2]);
	//char *mode = argv[3];
	//printf("nTotalElements: %d\nh: %d\n", nTotalElements, h);

	// initialize Input vector
	static float Input[MAX_SIZE]; // max_size

	int inputSize = 0;

	for (int i = 0; i < nTotalElements; i++)
	{
		int a = rand(); // Returns a pseudo-random integer
						// between 0 and RAND_MAX.
		int b = rand(); // same as above

		// float v = abs((int)(a * 100.0 + b) % 100);

		// inserir o valor v na posição i
		// Input[i] = v;
		Input[i] = a;
	}
	inputSize = nTotalElements;

	float nMax = FLT_MIN;
	float nMin = FLT_MAX;

	for (int i = 0; i < nTotalElements; i++)
	{
		if (Input[i] > nMax)
			nMax = Input[i];
		if (Input[i] < nMin)
			nMin = Input[i];
	}
	printf("nMin: %f\nnMax: %f\n", nMin, nMax);
	printf("Largura da faixa: %d", (int)(nMax - nMin)/h);

	unsigned int *histogram = (unsigned int *)malloc(sizeof(unsigned int) * h);

	// passar no vetor heap vazio inserindo os primeiros k valores do vetor de float, e a posição

	//__device__ Pair heap[140][MAX_HEAP_SIZE];   // aloca estático 140 heaps na global memory da gpu
	//__device__ int heapSize = k;

	/*
		Pair heap[140][MAX_HEAP_SIZE];
		int heapSize = 0;

		Pair temp;
		for (int j = 0; j < 140; j++)
		{
			for (int i = 0; i < k; i++)
			{
				//  printf("(%d, %d) ", i, temp.val);
				temp.key = Input[k * j + i];
				temp.val = k * j + i;
				// if (j == 2){
				// printf("\n%f\n", temp.key);
				// printf("\n%d\n", temp.val);
				// exit(0);
				//}
				insert(heap[j], &heapSize, temp);
			}
			heapSize = 0;
			// printf("\n%d\n", j);
		}
		heapSize = k;

		// dividir (por exemplo) em 2 blocos de threads que recebem 1000 sobra o resto pra fica fazendo decrease
		// entre cada bloco (cada bloco faz um decrease pra um valor)
		// no final esses 2 blocos de threads metade dos kMenores fica em um e metade no outro (não?)

		// algumas leem algumas inserem

		// int heap[MAX_HEAP_SIZE] = {60, 50, 40, 30, 20, 10}; // {70, 40, 50, 30, 20, 10};
		// int heapSize = 6;
	*/
	static float *cudaInput;
	static unsigned int *cudaHistogram_GH;
	static unsigned int(*cudaHistogram_BH)[(48 * 1024) / 4] = nullptr;
	// static Pair **cudaHeap;

	hipError_t err;

	err = hipMalloc((void **)&cudaInput, MAX_SIZE * sizeof(float));
	if (err != hipSuccess)
	{
		printf("%s in %s at line %d\n", hipGetErrorString(err), __FILE__, __LINE__);
		exit(EXIT_FAILURE);
	}
	err = hipMalloc((void **)&cudaHistogram_BH, sizeof(unsigned int) * (((48 * 1024) / 4) * 56));
	if (err != hipSuccess)
	{
		printf("%s in %s at line %d\n", hipGetErrorString(err), __FILE__, __LINE__);
		exit(EXIT_FAILURE);
	}
	err = hipMalloc((void **)&cudaHistogram_GH, sizeof(unsigned int) * h);
	if (err != hipSuccess)
	{
		printf("%s in %s at line %d\n", hipGetErrorString(err), __FILE__, __LINE__);
		exit(EXIT_FAILURE);
	}

	hipMemcpy(cudaInput, Input, inputSize * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(cudaHistogram_BH, histogram, sizeof(unsigned int) * (((48 * 1024) / 4) * 56), hipMemcpyHostToDevice);
	hipMemcpy(cudaHistogram_GH, histogram, sizeof(unsigned int) * h, hipMemcpyHostToDevice);

	/*
		int *mutex;
		hipMalloc(&mutex, sizeof(int));
		hipMemset(mutex, 0, sizeof(int));
	*/
	// Launch the Vector Add CUDA Kernel
	// int threadsPerBlock = 320;
	// int blocksPerGrid = (nTotalElements + threadsPerBlock - 1) / threadsPerBlock;
	//int blocks = 140;
	// int threadsPerBlock = (inputSize/blocks) + 1;
	//dim3 gridSize(56, 1, 1);
	//dim3 blockSize((1024), 1, 1);
	//printf("inputSize: %d\n", inputSize);
	//printf("CUDA kernel launch with %d blocks of %d threads\n", blocks, 320);

	chronometer_t chrono_va; // cria um chonometro para medir vectorAdd

	if (!strcmp(argv[3], "-gh"))
	{
		chrono_reset(&chrono_va);
		chrono_start(&chrono_va);
		globalHisto_SM<<<56, 1024>>>(cudaHistogram_GH, h, cudaInput, nTotalElements, nMin, nMax);
		hipDeviceSynchronize();
		chrono_stop(&chrono_va);
		hipMemcpy(histogram, cudaHistogram_GH, sizeof(unsigned int) * h, hipMemcpyDeviceToHost);
		chrono_reportTime(&chrono_va, (char *)"globalHisto_SM");
	}

	if (!strcmp(argv[3], "-bh"))
	{
		chrono_reset(&chrono_va);
		chrono_start(&chrono_va);

		// achar_K_Menores<<<gridSize, blockSize>>>(cudaHeap, cudaInput, heapSize, inputSize, mutex);

		// dim3 gridSize2(70, 1, 1);
		// dim3 blockSize2((320), 1, 1);
		// merge_heaps<<<gridSize2, blockSize2>>>(cudaHeap, heapSize, 70, mutex);

		// dim3 gridSize3(35, 1, 1);
		// dim3 blockSize3((320), 1, 1);
		// merge_heaps<<<gridSize3, blockSize3>>>(cudaHeap, heapSize, 35, mutex);

		// globalHisto_GM<<<56, 1024>>>(cudaHistogram, h, cudaInput, nTotalElements, nMin, nMax);
		blockHisto_SM<<<56, 1024>>>(cudaHistogram_BH, h, cudaInput, nTotalElements, nMin, nMax);

		// hipDeviceSynchronize();
		// chrono_stop(&chrono_va);

		// chrono_start(&chrono_va);

		// for (int i = 1; i <= 35; i++)
		//{
		//	for (int j = 0; j < heapSize; j++)
		//	{
		//		temp.key = heap[i][j].key;
		//		temp.val = heap[i][j].val;
		//		decreaseMax_seq(heap[0], heapSize, temp);
		//	}
		// }
		hipDeviceSynchronize();
		chrono_stop(&chrono_va);
		hipMemcpy(histogram, cudaHistogram_BH, sizeof(unsigned int) * h, hipMemcpyDeviceToHost);
		chrono_reportTime(&chrono_va, (char *)"blockHisto_SM");
	}

	if (!strcmp(argv[3], "-ghgm"))
	{
		chrono_reset(&chrono_va);
		chrono_start(&chrono_va);
		globalHisto_GM<<<56, 1024>>>(cudaHistogram_GH, h, cudaInput, nTotalElements, nMin, nMax);
		hipDeviceSynchronize();
		chrono_stop(&chrono_va);
		hipMemcpy(histogram, cudaHistogram_GH, sizeof(unsigned int) * h, hipMemcpyDeviceToHost);
		chrono_reportTime(&chrono_va, (char *)"globalHisto_GM");
	}

	//   printf("\n----- reportando o tempo total para as %d ativações do kernel -------",
	//		   N_REPETICOES);
	//chrono_reportTime(&chrono_va, (char *)"achar_K_Menores");

	//	printf("\n----- tempo médio por ativação do kernel -------");
	//	chrono_report_TimeInLoop(&chrono_va,
	//							(char *)"vectorAdd_kernel",
	//							 N_REPETICOES);
	printf("\n\n");

	// hipMemcpy(heap, cudaHeap, k * 140 * sizeof(Pair), hipMemcpyDeviceToHost);

	// int temp2 = -999999;
	//	for (int i = 0; i < heapSize; i++){
	//		printf("{%f, %d}, ", heap[0][i].key, heap[0][i].val);
	//	}

	err = hipGetLastError();

	if (err != hipSuccess)
	{
		fprintf(stderr, "Failed to launch vectorAdd kernel (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}

	double total_time_in_milliseconds = (double)chrono_gettotal(&chrono_va) /
										((double)1000 * 1000);
	printf("total_time_in_milliseconds: %lf ms\n", total_time_in_milliseconds);

	double OPS = ((double)nTotalElements * 1000) / total_time_in_milliseconds;
	printf("Throughput: %lf OP/s\n", OPS);
	printf("Throughput_ em GOP/ms: %lf GOP/ms\n", OPS / (1000 * 1000 * 1000));
	//printf("Vazão de dados em GB/s: %lf\n", (nTotalElements * 4) / (total_time_in_milliseconds / 1000));

//	for (int i = 0; i < h; i++)
//		printf("%ui ", histogram[i]);

	// verifyOutput(NULL, NULL, 0, 0);

	/*for (int i = 0; i < inputSize; i++)
		cudaInput[i] = input[i];
	for (int i = 0; i < k; i++)
		cudaHeap[i] = heap[i];*/

	// int heap[MAX_HEAP_SIZE] = {60, 50, 40, 30, 20, 10}; // {70, 40, 50, 30, 20, 10};
	// int heapSize = 6;

	return 0;
}
