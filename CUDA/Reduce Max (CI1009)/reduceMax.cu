#include "hip/hip_runtime.h"
// FEITO POR:
// André Miguel Sikora Marques
// GRR20182593

#include <stdlib.h>
#include <stdio.h>
#include <math.h>
#include <float.h>

#include "chrono.c"

#include "hip/hip_runtime_api.h" // mudado aqui por WZ

// para compilar:
// nvcc -arch sm_61 --std=c++14 -o trab3 reduceMax.cu

#define MAX_SIZE ((256 * 1024 * 1024) + 4)
// #define MAX_SIZE 268435460

__device__ static float atomicMax(float *address, float val)
{
	int *address_as_i = (int *)address;
	int old = *address_as_i, assumed;
	do
	{
		assumed = old;
		old = ::atomicCAS(address_as_i, assumed,
						  __float_as_int(::fmaxf(val, __int_as_float(assumed))));
	} while (assumed != old);
	return __int_as_float(old);
}

__device__ float global_max;

__global__ void
reduceMax_coalesced(float *max, float *Input, unsigned nElements)
{
	int thread_index = threadIdx.x + blockIdx.x * blockDim.x;

	if (thread_index == 0)
		global_max = FLT_MIN;

	int increment = blockDim.x * gridDim.x;

	float max_local = FLT_MIN;
	for (int i = thread_index; i < nElements; i += increment)
		if (Input[i] > max_local)
			max_local = Input[i];

	__shared__ unsigned int shared_max[(48 * 1024) / 4];
	if (thread_index < (48 * 1024) / 4)
		shared_max[thread_index] = max_local;
	__syncthreads();

	for (unsigned int stride = blockDim.x; stride > 0; stride /= 2)
	{
		__syncthreads();
		if (thread_index < stride)
			if (shared_max[thread_index] < shared_max[thread_index + stride])
				shared_max[thread_index] = shared_max[thread_index + stride];
	}

	__syncthreads();

	atomicMax(&global_max, shared_max[0]);

	/*
	max_local = shared_max[0];
	for (int i = 1; i < blockDim.x; i++)
		if (shared_max[i] > max_local)
			max_local = shared_max[i];
	*/

	__syncthreads();

	if (thread_index == 0)
		*max = global_max;

}

__global__ void
reduceMax_blockCoalesced(float *max, float *Input, unsigned nElements)
{
	int thread_index = threadIdx.x + blockIdx.x * blockDim.x;

	if (thread_index == 0)
		global_max = FLT_MIN;

	// int increment = blockDim.x * gridDim.x;
	int increment = blockDim.x;

	unsigned int nElements_block = nElements / gridDim.x;

	unsigned int begin = min(nElements_block * blockIdx.x, nElements);
	unsigned int end = min(begin + nElements_block, nElements);

	float max_local = FLT_MIN;
	for (int i = begin; i < end; i += increment)
		if (Input[i] > max_local)
			max_local = Input[i];

	__shared__ unsigned int shared_max[(48 * 1024) / 4];
	if (thread_index < (48 * 1024) / 4)
		shared_max[thread_index] = max_local;
	__syncthreads();

	for (unsigned int stride = blockDim.x; stride > 0; stride /= 2)
	{
		__syncthreads();
		if (thread_index < stride)
			if (shared_max[thread_index] < shared_max[thread_index + stride])
				shared_max[thread_index] = shared_max[thread_index + stride];
	}

	/*
	for (unsigned int stride = 1; stride <= blockDim.x; stride *= 2)
	{
		__syncthreads();
		if (thread_index % stride == 0)
			if (shared_max[thread_index] < shared_max[thread_index + stride])
				shared_max[thread_index] = shared_max[thread_index + stride];
	}
	*/

	__syncthreads();

	atomicMax(&global_max, shared_max[0]);

	/*
	max_local = shared_max[0];
	for (int i = 1; i < blockDim.x; i++)
		if (shared_max[i] > max_local)
			max_local = shared_max[i];
	*/

	__syncthreads();

	if (thread_index == 0)
		max[0] = global_max;
}

//////////////
int main(int argc, char *argv[])
{

	if (argc < 3)
	{
		fprintf(stderr, "usage:\n\t./trab2 <nTotalElements> <nR>\n");
		exit(-1);
	}

	int nTotalElements = atoi(argv[1]);
	int nR = atoi(argv[2]);

	// initialize Input vector
	static float Input[MAX_SIZE]; // max_size

	int inputSize = 0;

	for (int i = 0; i < nTotalElements; i++)
	{
		int a = rand(); // Returns a pseudo-random integer
						// between 0 and RAND_MAX.
		int b = rand(); // same as above

		// float v = abs((int)(a * 100.0 + b) % 100);

		// inserir o valor v na posição i
		// Input[i] = v;
		Input[i] = a;
	}
	inputSize = nTotalElements;

	static float *cudaInput;
	static float *cudaMax;

	hipError_t err;

	err = hipMalloc((void **)&cudaInput, MAX_SIZE * sizeof(float));
	if (err != hipSuccess)
	{
		printf("%s in %s at line %d\n", hipGetErrorString(err), __FILE__, __LINE__);
		exit(EXIT_FAILURE);
	}
	err = hipMalloc((void **)&cudaMax, sizeof(float));
	if (err != hipSuccess)
	{
		printf("%s in %s at line %d\n", hipGetErrorString(err), __FILE__, __LINE__);
		exit(EXIT_FAILURE);
	}

	float max[1] = {FLT_MIN};
	hipMemcpy(cudaInput, Input, inputSize * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(cudaMax, max, sizeof(float), hipMemcpyHostToDevice);

	printf("\n\n");

	chronometer_t chrono_va; // cria um chonometro para medir vectorAdd

	long long time = 0;
	for(int i = 0; i < nR; i++){
		chrono_reset(&chrono_va);
		chrono_start(&chrono_va);
		reduceMax_coalesced<<<56, 1024>>>(cudaMax, cudaInput, nTotalElements);
		hipDeviceSynchronize();
		chrono_stop(&chrono_va);
		time += chrono_gettotal(&chrono_va);
	}
	double avg_ms_coalesced = (double)time/(nR * 1000);
	printf("avg_time_coalesced: %f microseconds\n", avg_ms_coalesced);

	double OPS = ((double)nTotalElements * 1000 * 1000) / (avg_ms_coalesced * (1000 * 1000 * 1000));
	printf("throughput_coalesced: %lf GOP/s\n\n", OPS);

	max[0] = FLT_MIN;
	hipMemcpy(cudaMax, max, sizeof(float), hipMemcpyHostToDevice);
	time = 0;
	for(int i = 0; i < nR; i++){
		chrono_reset(&chrono_va);
		chrono_start(&chrono_va);
		reduceMax_blockCoalesced<<<56, 1024>>>(cudaMax, cudaInput, nTotalElements);
		hipDeviceSynchronize();
		chrono_stop(&chrono_va);
		time += chrono_gettotal(&chrono_va);
	}
	double avg_ms_blockCoalesced = (double)time/(nR * 1000);
	printf("avg_time_blockCoalesced: %f microseconds\n", avg_ms_blockCoalesced);

	OPS = ((double)nTotalElements * 1000 * 1000) / (avg_ms_blockCoalesced * (1000 * 1000 * 1000));
	printf("throughput_blockCoalesced: %lf GOP/s\n", OPS);

	printf("\n\n");

	double speedup = avg_ms_coalesced / avg_ms_blockCoalesced;
	printf("speedup: %f\n", speedup);

	printf("\n\n");

	err = hipGetLastError();

	if (err != hipSuccess)
	{
		fprintf(stderr, "Failed to launch reduceMax kernel (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}

	//double total_time_in_milliseconds = (double)chrono_gettotal(&chrono_va) /
	//									((double)1000 * 1000);
	//printf("total_time_in_milliseconds: %lf ms\n", total_time_in_milliseconds);

	return 0;
}
