#include "hip/hip_runtime.h"
// FEITO POR:
// André Miguel Sikora Marques
// GRR20182593

#include <stdlib.h>
#include <stdio.h>
#include <math.h>

#include "chrono.c"

#include "hip/hip_runtime_api.h" // mudado aqui por WZ
/**
 * CUDA Kernel Device code
 *
 * Computes the vector addition of A and B into C. The 3 vectors have the same
 * number of elements numElements.
 */

// (c) by W.Zola set/23
//
// EXAMPLE Sequential MAX-Heap,
//				decreaseMax operation
//				and others

// para compilar:
// gcc -O3 max-heap.c -o max-heap -lm

// #define SHOW_DECREASE_MAX_STEPS 1

#define MAX_HEAP_SIZE (2048)
#define MAX_SIZE 20000000

typedef struct
{
	float key; // inserir um valor v float na chave
			   // (obtido do vetor de entrada Input)
	int val;   // inserir a posiçao p como valor val
} Pair;

__device__ void lock(int *mutex)
{
	while (atomicCAS(mutex, 0, 1) != 0)
		;
}
__device__ void unlock(int *mutex)
{
	atomicExch(mutex, 0);
}

__device__ void drawHeapTree(Pair heap[], int size, int nLevels) // FIX ME!
{
	int offset = 0;
	int space = (int)pow(2, nLevels - 1);
	// int space = 0;

	int nElements = 1;
	for (int level = 0; level < nLevels; level++)
	{

		// print all elements in this level
		for (int i = offset; i < size && i < (offset + nElements); i++)
		{
			printf("[%3lf, %d]", heap[i].key, heap[i].val);
		}
		printf("\n");

		offset += nElements;
		space = nElements - 1;
		nElements *= 2;
	}
}

__device__ void swap(Pair *a, Pair *b) //__attribute__((always_inline));
{
	Pair temp = *a;
	*a = *b;
	*b = temp;
}

void maxHeapify_seq(Pair heap[], int size, int i)
{
	while (1)
	{
		int largest = i;
		int left = 2 * i + 1;
		int right = 2 * i + 2;

		if (left < size && heap[left].key > heap[largest].key)
			largest = left;

		if (right < size && heap[right].key > heap[largest].key)
			largest = right;

		if (largest != i)
		{
			// swap(&heap[i], &heap[largest]); // Using the swap function
			Pair temp = heap[i];
			heap[i] = heap[largest];
			heap[largest] = temp;
			i = largest;
		}
		else
		{
			break;
		}
	}
}

// #define parent(pos) ( pos/2 ) // SE nao usar posicao 0
#define parent(pos) ((pos - 1) / 2)

void heapifyUp(Pair heap[], int *size, int pos, int old_pos)
{
	int val = heap[pos].key;
	while (pos > 0 && val > heap[parent(pos)].key)
	{
		heap[pos] = heap[parent(pos)];
		pos = parent(pos);
	}
	heap[pos].key = val;
	heap[pos].val = old_pos;
}

void insert(Pair heap[], int *size, Pair element)
{
	*size += 1;
	int last = *size - 1;

	heap[last] = element;
	// printf("before heapifyUp ");
	heapifyUp(heap, size, last, element.val);
	// printf("after heapifyUp ");
}

int isMaxHeap(Pair heap[], int size)
{
	for (int i = 1; i < size; i++)
		if (heap[i].key <= heap[parent(i)].key)
			continue;
		else
		{
			printf("\nbroke at [%d]=%lf\n", i, heap[i].key);
			printf("father at [%d]=%lf\n",
				   parent(i), heap[parent(i)].key);
			return 0;
		}
	return 1;
}

void decreaseMax_seq(Pair heap[], int size, Pair new_value)
{
	if (size == 0) // Heap is empty
		return;

	if (heap[0].key > new_value.key)
	{
		heap[0] = new_value;
#if SHOW_DECREASE_MAX_STEPS
		drawHeapTree(heap, size, 4);
		printf("	~~~~~~~~~~~~~~~~~~~~~~~~~\n");
#endif
		maxHeapify_seq(heap, size, 0);
	}
}

__device__ void maxHeapify(Pair heap[], int size, int i)
{
	while (1)
	{
		int largest = i;
		int left = 2 * i + 1;
		int right = 2 * i + 2;

		if (left < size && heap[left].key > heap[largest].key)
			largest = left;

		if (right < size && heap[right].key > heap[largest].key)
			largest = right;

		if (largest != i)
		{
			// swap(&heap[i], &heap[largest]); // Using the swap function
			Pair temp = heap[i];
			heap[i] = heap[largest];
			heap[largest] = temp;
			i = largest;
		}
		else
		{
			break;
		}
	}
}

__device__ void decreaseMax(Pair heap[], int size, Pair new_value)
{
	if (size == 0) // Heap is empty
		return;

	if (heap[0].key > new_value.key)
	{
		heap[0] = new_value;
#if SHOW_DECREASE_MAX_STEPS
		drawHeapTree(heap, size, 4);
		printf("	~~~~~~~~~~~~~~~~~~~~~~~~~\n");
#endif
		maxHeapify(heap, size, 0);
	}
}

__global__ void
achar_K_Menores(Pair (*heap)[MAX_HEAP_SIZE], float *input, int heapSize, int inputSize, int *mutex)
{
	int thread_index = threadIdx.x + blockIdx.x * blockDim.x;
	int thread_index_next = (threadIdx.x + 1) + blockIdx.x * blockDim.x;

	int index = (thread_index) * (inputSize / (320 * 140));
	int next_index = (thread_index_next) * (inputSize / (320 * 140));

	Pair temp;
	if (index < inputSize)
	{
		for (int i = index; i < next_index; i++)
		{
			if (i < inputSize)
			{
				temp.key = input[i];
				temp.val = i;
				bool leave = true;
				while (leave)
				{
					if (atomicCAS(mutex, 0, 1) == 0)
					{
						decreaseMax(heap[blockIdx.x], heapSize, temp);
						leave = false;
						atomicExch(mutex, 0);
						break;
					}
					break;
					// decreaseMax(heap[blockIdx.x], heapSize, temp);
				}
			}
			else
				break;
		}
		// printf("%d is legal\n", index);
		// if(blockIdx.x >= 140)
		//	printf("blockId: %d\n", blockIdx.x);
		// temp.key = input[index];
		// temp.val = index;
		// decreaseMax(heap[blockIdx.x], heapSize, temp);
	}
}

__global__ void
merge_heaps(Pair (*heap)[2048], int heapSize, int nBlocks, int *mutex)
{
	// int index = (threadIdx.x + blockIdx.x * blockDim.x) * 320;
	// int next_index = (threadIdx.x + (blockIdx.x + 1) * blockDim.x) * 320;

	Pair temp;
	// if (index < heapSize){
	for (int i = 0; i < heapSize; i++)
	{
		bool leave = true;
		while (leave)
		{
			if (atomicCAS(mutex, 0, 1) == 0)
			{
				temp.key = heap[blockIdx.x + nBlocks][i].key;
				temp.val = heap[blockIdx.x + nBlocks][i].val;
				decreaseMax(heap[blockIdx.x], heapSize, temp);
				leave = false;
				atomicExch(mutex, 0);
				break;
			}
			break;
		}
	}
	//}
}

void verifyOutput(const float *Input,
				  const Pair *Output, // pair_t é o tipo de um par (v,p)
				  int nTotalElmts,
				  int k)
{
	// codigo da verificacao a ser incluido por voce
	// voce deve verificar se o conjunto de pares de saida está correto
	// e imprimir uma das mensagens abaixo
	int ok = 1;

	// inserir aqui o codigo da verificacao
	// uma implementação possível para o verificador seria
	// (nao precisa seguir essa descrição, voce pode fazer outro método
	//  de verificação)
	//
	// 1) Criar um vetor I de pares (chave, valor)
	//	os os elementos de I devem ser copias
	//	de cada valor (e,p) do vetor de entrada Input
	//	(ou seja, cada valor e que veio da posição p da entrada)
	// 2) Ordenar o vetor I em ordem crescente,
	//	obtendo-se um outro vetor Is (ordenado em ordem crescente de chaves)
	//	usando um algoritmo de ordenação do tipo (chave, valor)
	//	(por exemplo ordenação da stdlib, caso exista)
	// 3) Para cada par (ki,vi) pertencente ao vetor Output
	//	  procurar a chave ki dentre K primeiros elementos
	//	  de Is.
	//	  Se a chave Ki estiver em Is com valor val==vi continue
	//	  senão faça ok = 0 e reporte o erro abaixo

	if (ok)
		printf("\nOutput set verified correctly.\n");
	else
		printf("\nOutput set DID NOT compute correctly!!!\n");
}

//////////////
int main(int argc, char *argv[])
{
	int nTotalElements = atoi(argv[1]);
	int k = atoi(argv[2]);

	// initialize Input vector
	static float Input[MAX_SIZE]; // max_size

	int inputSize = 0;

	for (int i = 0; i < nTotalElements; i++)
	{
		int a = rand(); // Returns a pseudo-random integer
						// between 0 and RAND_MAX.
		int b = rand(); // same as above

		// float v = abs((int)(a * 100.0 + b) % 100);

		// inserir o valor v na posição i
		// Input[i] = v;
		Input[i] = a;
	}
	inputSize = nTotalElements;

	// passar no vetor heap vazio inserindo os primeiros k valores do vetor de float, e a posição

	//__device__ Pair heap[140][MAX_HEAP_SIZE];   // aloca estático 140 heaps na global memory da gpu
	//__device__ int heapSize = k;

	Pair heap[140][MAX_HEAP_SIZE];
	int heapSize = 0;

	Pair temp;
	for (int j = 0; j < 140; j++)
	{
		for (int i = 0; i < k; i++)
		{
			//  printf("(%d, %d) ", i, temp.val);
			temp.key = Input[k * j + i];
			temp.val = k * j + i;
			// if (j == 2){
			// printf("\n%f\n", temp.key);
			// printf("\n%d\n", temp.val);
			// exit(0);
			//}
			insert(heap[j], &heapSize, temp);
		}
		heapSize = 0;
		// printf("\n%d\n", j);
	}
	heapSize = k;

	// dividir (por exemplo) em 2 blocos de threads que recebem 1000 sobra o resto pra fica fazendo decrease
	// entre cada bloco (cada bloco faz um decrease pra um valor)
	// no final esses 2 blocos de threads metade dos kMenores fica em um e metade no outro (não?)

	// algumas leem algumas inserem

	// int heap[MAX_HEAP_SIZE] = {60, 50, 40, 30, 20, 10}; // {70, 40, 50, 30, 20, 10};
	// int heapSize = 6;

	static float *cudaInput;
	// static Pair **cudaHeap;
	Pair(*cudaHeap)[MAX_HEAP_SIZE] = nullptr;

	hipError_t err;

	err = hipMalloc((void **)&cudaInput, MAX_SIZE * sizeof(float));
	if (err != hipSuccess)
	{
		printf("%s in %s at line %d\n", hipGetErrorString(err), __FILE__, __LINE__);
		exit(EXIT_FAILURE);
	}
	err = hipMalloc((void **)&cudaHeap, MAX_HEAP_SIZE * sizeof(Pair) * 140);
	if (err != hipSuccess)
	{
		printf("%s in %s at line %d\n", hipGetErrorString(err), __FILE__, __LINE__);
		exit(EXIT_FAILURE);
	}

	hipMemcpy(cudaInput, Input, inputSize * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(cudaHeap, heap, k * 140 * sizeof(Pair), hipMemcpyHostToDevice);

	int *mutex;
	hipMalloc(&mutex, sizeof(int));
	hipMemset(mutex, 0, sizeof(int));

	// Launch the Vector Add CUDA Kernel
	// int threadsPerBlock = 320;
	// int blocksPerGrid = (nTotalElements + threadsPerBlock - 1) / threadsPerBlock;
	int blocks = 140;
	// int threadsPerBlock = (inputSize/blocks) + 1;
	dim3 gridSize(140, 1, 1);
	dim3 blockSize((320), 1, 1);
	printf("inputSize: %d\n", inputSize);
	printf("CUDA kernel launch with %d blocks of %d threads\n", blocks, 320);

	chronometer_t chrono_va; // cria um chonometro para medir vectorAdd

	chrono_start(&chrono_va);

	achar_K_Menores<<<gridSize, blockSize>>>(cudaHeap, cudaInput, heapSize, inputSize, mutex);

	dim3 gridSize2(70, 1, 1);
	dim3 blockSize2((320), 1, 1);
	merge_heaps<<<gridSize2, blockSize2>>>(cudaHeap, heapSize, 70, mutex);

	dim3 gridSize3(35, 1, 1);
	dim3 blockSize3((320), 1, 1);
	merge_heaps<<<gridSize3, blockSize3>>>(cudaHeap, heapSize, 35, mutex);

	// hipDeviceSynchronize();
	// chrono_stop(&chrono_va);

	hipMemcpy(heap, cudaHeap, k * 140 * sizeof(Pair), hipMemcpyDeviceToHost);

	// chrono_start(&chrono_va);

	for (int i = 1; i <= 35; i++)
	{
		for (int j = 0; j < heapSize; j++)
		{
			temp.key = heap[i][j].key;
			temp.val = heap[i][j].val;
			decreaseMax_seq(heap[0], heapSize, temp);
		}
	}
	hipDeviceSynchronize();
	chrono_stop(&chrono_va);

	//   printf("\n----- reportando o tempo total para as %d ativações do kernel -------",
	//		   N_REPETICOES);
	chrono_reportTime(&chrono_va, (char *)"achar_K_Menores");

	//	printf("\n----- tempo médio por ativação do kernel -------");
	//	chrono_report_TimeInLoop(&chrono_va,
	//							(char *)"vectorAdd_kernel",
	//							 N_REPETICOES);
	printf("\n\n");

	hipMemcpy(heap, cudaHeap, k * 140 * sizeof(Pair), hipMemcpyDeviceToHost);

	// int temp2 = -999999;
	//	for (int i = 0; i < heapSize; i++){
	//		printf("{%f, %d}, ", heap[0][i].key, heap[0][i].val);
	//	}

	err = hipGetLastError();

	if (err != hipSuccess)
	{
		fprintf(stderr, "Failed to launch vectorAdd kernel (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}

	double total_time_in_milliseconds = (double)chrono_gettotal(&chrono_va) /
										((double)1000 * 1000);
	printf("total_time_in_milliseconds: %lf ms\n", total_time_in_milliseconds);

	double OPS = (nTotalElements) / total_time_in_milliseconds;
	printf("Throughput: %lf OP/ms\n", OPS);
	printf("Throughput em MOPS: %lf MOP/ms\n", OPS / 1000000);

	verifyOutput(NULL, NULL, 0, 0);

	/*for (int i = 0; i < inputSize; i++)
		cudaInput[i] = input[i];
	for (int i = 0; i < k; i++)
		cudaHeap[i] = heap[i];*/

	// int heap[MAX_HEAP_SIZE] = {60, 50, 40, 30, 20, 10}; // {70, 40, 50, 30, 20, 10};
	// int heapSize = 6;

	return 0;
}
